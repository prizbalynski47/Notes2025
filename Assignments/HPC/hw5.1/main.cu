#include "kernel.cu"
#include <stdio.h>

// Establish a few constants
#define WIDTH 1024
#define HEIGHT 1024
#define CHANNELS 3 // RGB

int main() {
    const int img_size_rgb = WIDTH * HEIGHT * CHANNELS;
    const int img_size_grey = WIDTH * HEIGHT;

    unsigned char* h_rgb = new unsigned char[img_size_rgb];
    unsigned char* h_grey = new unsigned char[img_size_grey];

    // Read in the file
    std::ifstream input_file("gc_conv_1024x1024.raw", std::ios::binary);
    if (!input_file) {
        std::cerr << "Error opening input file.\n";
        return 1;
    }
    input_file.read(reinterpret_cast<char*>(h_rgb), img_size_rgb);
    input_file.close();

    // Allocate device variables
    unsigned char* d_rgb;
    unsigned char* d_grey;
    hipMalloc(&d_rgb, img_size_rgb);
    hipMalloc(&d_grey, img_size_grey);

    // Copy host variables to device
    hipMemcpy(d_rgb, h_rgb, img_size_rgb, hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    // Start the kernal
    int threads_per_block = 256;
    int num_blocks = (WIDTH * HEIGHT + threads_per_block - 1) / threads_per_block;
    rgb_to_greyscale<<<num_blocks, threads_per_block>>>(d_rgb, d_grey, WIDTH, HEIGHT);

    // Check if the kernal worked
    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) printf("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // copy result back
    hipMemcpy(h_grey, d_grey, img_size_grey, hipMemcpyDeviceToHost);

    // Write output file
    std::ofstream output_file("gc.raw", std::ios::binary);
    if (!output_file) {
        std::cerr << "Error opening output file.\n";
        return 1;
    }
    output_file.write(reinterpret_cast<char*>(h_grey), img_size_grey);
    output_file.close();

    // Free the memory
    delete[] h_rgb;
    delete[] h_grey;
    hipFree(d_rgb);
    hipFree(d_grey);

    // Send it on home
    return 0
}
