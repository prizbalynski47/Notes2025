#include "hip/hip_runtime.h"
__global__
void rgb_to_greyscale(unsigned char* input, unsigned char* output, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = width * height;

    if (idx < total_pixels) {
        int rgb_idx = idx * CHANNELS;
        unsigned char r = input[rgb_idx];
        unsigned char g = input[rgb_idx + 1];
        unsigned char b = input[rgb_idx + 2];

        // Standard luminance formula
        // Found at https://www.w3.org/TR/AERT/#color-contrast
        output[idx] = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
    }
}