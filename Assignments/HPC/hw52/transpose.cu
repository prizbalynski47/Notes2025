#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <chrono>

// Define some constants. In some more general applications we might want to make the width and
// height parameters or grab them from the file, but for this assignment we're just using the one image.
#define WIDTH 1024
#define HEIGHT 1024
#define CHANNELS 3
#define TILE_DIM 32

using namespace std;

__global__ void transpose_global(unsigned char* in, unsigned char* out, int width, int height) {
    // Assigin each thread to a pixel location
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Compute transposed postion and store it in the out array
    if (x < width && y < height) {
        for (int c = 0; c < CHANNELS; ++c) {
            out[(x * height + y) * CHANNELS + c] = in[(y * width + x) * CHANNELS + c];
        }
    }
}

__global__ void transpose_shared(unsigned char* in, unsigned char* out, int width, int height) {
    // Make shared memory tile
    __shared__ unsigned char tile[TILE_DIM][TILE_DIM * CHANNELS];

    // Get the global x and y locations for the thread
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;

    // Copy from global memory into the tile
    if (x < width && y < height) {
        for (int c = 0; c < CHANNELS; ++c)
            tile[threadIdx.y][threadIdx.x * CHANNELS + c] = in[(y * width + x) * CHANNELS + c];
    }

    // Wait for everything to load their tile
    __syncthreads();

    // Transpose indeces within the block
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    // Copy back from shared to global at new transposed locations
    if (x < height && y < width) {
        for (int c = 0; c < CHANNELS; ++c)
            out[(y * height + x) * CHANNELS + c] = tile[threadIdx.x][threadIdx.y * CHANNELS + c];
    }
}

// Transposes the image serially for validation
void transpose_cpu(unsigned char* in, unsigned char* out, int width, int height) {
    for (int y = 0; y < height; ++y)
        for (int x = 0; x < width; ++x)
            for (int c = 0; c < CHANNELS; ++c)
                out[(x * height + y) * CHANNELS + c] = in[(y * width + x) * CHANNELS + c];
}

// Simple validation that goes pixel by pixel and checks they match
bool validate(unsigned char* a, unsigned char* b, int size) {
    for (int i = 0; i < size; ++i)
        if (a[i] != b[i])
            return false;
    return true;
}

// Standard code to write to file
void save_to_file(const string& filename, unsigned char* data, int size) {
    ofstream out(filename, ios::binary);
    out.write(reinterpret_cast<char*>(data), size);
    out.close();
}

int main() {
    // Calculate the character size of the image
    const int imageSize = WIDTH * HEIGHT * CHANNELS;

    // Make the containers for the image data
    unsigned char* h_input = new unsigned char[imageSize];
    unsigned char* h_output_global = new unsigned char[imageSize];
    unsigned char* h_output_shared = new unsigned char[imageSize];
    unsigned char* h_reference = new unsigned char[imageSize];

    // Read in the file
    ifstream in("gc_1024x1024.raw", ios::binary);
    if (!in) {
        cerr << "Failed to open input file." << endl;
        return 1;
    }
    in.read(reinterpret_cast<char*>(h_input), imageSize);
    in.close();

    // Allocate GPU memory
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, imageSize);
    hipMalloc(&d_output, imageSize);

    hipMemcpy(d_input, h_input, imageSize, hipMemcpyHostToDevice);

    // Set up thread blocks
    dim3 blockSize(TILE_DIM, TILE_DIM);
    dim3 gridSize((WIDTH + TILE_DIM - 1) / TILE_DIM, (HEIGHT + TILE_DIM - 1) / TILE_DIM);

    // Global memory transpose (with timing)
    auto start = chrono::high_resolution_clock::now();
    transpose_global<<<gridSize, blockSize>>>(d_input, d_output, WIDTH, HEIGHT);
    hipDeviceSynchronize();
    auto end = chrono::high_resolution_clock::now();
    float time_global = chrono::duration<float, milli>(end - start).count();

    // Get the result back from the GPU
    hipMemcpy(h_output_global, d_output, imageSize, hipMemcpyDeviceToHost);

    // Shared memory transpose (also with timing)
    start = chrono::high_resolution_clock::now();
    transpose_shared<<<gridSize, blockSize>>>(d_input, d_output, WIDTH, HEIGHT);
    hipDeviceSynchronize();
    end = chrono::high_resolution_clock::now();
    float time_shared = chrono::duration<float, milli>(end - start).count();

    // Get the result back from the GPU
    hipMemcpy(h_output_shared, d_output, imageSize, hipMemcpyDeviceToHost);

    // CPU transpose for validation
    transpose_cpu(h_input, h_reference, WIDTH, HEIGHT);

    // Validate against CPU transpose
    bool valid_global = validate(h_output_global, h_reference, imageSize);
    bool valid_shared = validate(h_output_shared, h_reference, imageSize);

    // Report validation results
    cout << "Global valid: " << (valid_global ? "Yes" : "No") << endl;
    cout << "Shared valid: " << (valid_shared ? "Yes" : "No") << endl;

    // Calculate bandwidths
    float bandwidth_global = (float)imageSize / (time_global / 1000) / 1e9;
    float bandwidth_shared = (float)imageSize / (time_shared / 1000) / 1e9;

    // Report bandwidths
    // My bandwidth results on notchpeak:
    // Global transpose time: 2.40865 ms, Bandwidth: 1.30601 GB/s
    // Shared transpose time: 0.610389 ms, Bandwidth: 5.15364 GB/s
    cout << "Global transpose time: " << time_global << " ms, Bandwidth: " << bandwidth_global << " GB/s" << endl;
    cout << "Shared transpose time: " << time_shared << " ms, Bandwidth: " << bandwidth_shared << " GB/s" << endl;

    // Save the matrices to file
    save_to_file("transposed_global.raw", h_output_global, imageSize);
    save_to_file("transposed_shared.raw", h_output_shared, imageSize);

    // End of file resource cleanup
    hipFree(d_input);
    hipFree(d_output);
    delete[] h_input;
    delete[] h_output_global;
    delete[] h_output_shared;
    delete[] h_reference;

    return 0;
}
